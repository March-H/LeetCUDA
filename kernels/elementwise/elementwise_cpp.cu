#include "hip/hip_runtime.h"
#include "elementwise_cpp.h"

#define CHECK_TORCH_TENSOR_DTYPE(T, torch_type)                                  \
  if (((T).options().dtype() != (torch_type))) {                                 \
    std::cout << "Tensor Info:" << (T).options().dtype() << std::endl;           \
    throw std::runtime_error("value must be " #torch_type);                      \
  }

#define TORCH_BINDING_ELEM_ADD(packed_type, element_type)                                   \
  __global__ void elementwise_add_##packed_type(element_type *a,                            \
                                                element_type *b,                            \
                                                element_type *c, int N) {                   \
    elementwise_add_##packed_type##_kernel(a, b, c, N);                                     \
  }

TORCH_BINDING_ELEM_ADD(f32, float)
TORCH_BINDING_ELEM_ADD(f32x4, float)
TORCH_BINDING_ELEM_ADD(f16, half)
TORCH_BINDING_ELEM_ADD(f16x2, half)
TORCH_BINDING_ELEM_ADD(f16x8, half)
TORCH_BINDING_ELEM_ADD(f16x8_pack, half)

template<typename ElementType, torch::Dtype TorchType, int n_elements>
void elementwise_add(const torch::Tensor &a,
                     const torch::Tensor &b,
                     torch::Tensor &c,
                     void (*kernel)(ElementType *, ElementType *, ElementType *, int)
) {
    CHECK_TORCH_TENSOR_DTYPE(a, TorchType);
    CHECK_TORCH_TENSOR_DTYPE(b, TorchType);
    CHECK_TORCH_TENSOR_DTYPE(c, TorchType);
    const int ndim = a.dim();
    if (ndim != 2) {
        int N = 1;
        for (int i = 0; i < ndim; i++) {
            N *= a.size(i);
        }
        dim3 block(256 / (n_elements));
        dim3 grid((N + 256 - 1) / 256);
        kernel<<<grid, block>>>(
            reinterpret_cast<ElementType *>(a.data()),
            reinterpret_cast<ElementType *>(b.data()),
            reinterpret_cast<ElementType *>(c.data()),
            N);
    } else {
        const int S = a.size(0);
        const int K = a.size(1);
        const int N = S * K;
        if (K / n_elements <= 1024) {
            dim3 block(K / n_elements);
            dim3 grid(S);
            kernel<<<grid, block>>>(
                reinterpret_cast<ElementType *>(a.data()),
                reinterpret_cast<ElementType *>(b.data()),
                reinterpret_cast<ElementType *>(c.data()),
                N);
        } else {
            int N = 1;
            for (int i = 0; i < ndim; i++) {
                N *= a.size(i);
            }
            dim3 block(256 / (n_elements));
            dim3 grid((N + 256 - 1) / 256);
            kernel<<<grid, block>>>(
                reinterpret_cast<ElementType *>(a.data()),
                reinterpret_cast<ElementType *>(b.data()),
                reinterpret_cast<ElementType *>(c.data()),
                N);
        }
    }
}

int main() {
}
